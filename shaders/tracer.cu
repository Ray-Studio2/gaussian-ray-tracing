#include "hip/hip_runtime.h"
#include <optix.h>

#include <glm.hpp>
#include <gtc/matrix_transform.hpp>
#include <gtc/quaternion.hpp>

#include "tracer.cuh"

extern "C"
{
	#ifndef __PARAMS__
		__constant__ Params params;
	#define __PARAMS__ 1
	#endif
}

extern "C" __global__ void __raygen__raygeneration()
{
	// Initialize payload.
	RayPayload payload;
	payload.t_hit            = 0.0f;
	payload.numBounces       = 0;
	payload.hitNormal        = make_float3(0.0f);
	payload.accumColor       = make_float3(0.0f);
	payload.accumAlpha       = 0.0f;
	payload.blockingRadiance = 0.0f;
	payload.directLight      = make_float3(0.0f);

	RayData rayData;
	rayData.initialize();
	payload.rayData = &rayData;
	
	setNextTraceState(TraceGaussianPass);

	if (!params.mode_fisheye) {
		getRay(optixGetLaunchIndex(),
			   -params.U,
			   -params.V,
			   params.W,
			   params.eye,
			   params.width,
			   params.height,
			   payload.currRayOrigin,
			   payload.currRayDirection);
	}
	else {
		getFishEyeRay(optixGetLaunchIndex(),
					  -params.U,
					  -params.V,
					  params.W,
					  params.eye,
					  params.width,
					  params.height,
					  payload.currRayOrigin,
					  payload.currRayDirection);
	}

	unsigned int timeout = 0;
	while ((length(payload.currRayDirection) > 0.1f) && (payload.numBounces < MAX_BOUNCES)) {
		const float3 ray_o = payload.currRayOrigin;
		const float3 ray_d = payload.currRayDirection;

		traceMesh(ray_o, ray_d, &payload);
		
		if (getNextTraceState() == TraceTerminate) break;
		
		float4 gsRadDns;
		if (getNextTraceState() == TraceLastGaussianPass) {
			gsRadDns = traceGaussians(rayData, 
										ray_o, 
										ray_d, 
										params.t_min, 
										params.t_max, 
										&payload);
			float3 radiance = make_float3(gsRadDns.x, 
							  			  gsRadDns.y, 
										  gsRadDns.z);
			float alpha = gsRadDns.w;

			payload.directLight = radiance * alpha;
			payload.accumAlpha = clamp(payload.accumAlpha + alpha , 0.0f, 1.0f);
			setNextTraceState(TraceTerminate);
		}
		else {
			gsRadDns = traceGaussians(rayData, 
										ray_o, 
										ray_d, 
										params.t_min, 
										payload.t_hit, 
										&payload);
			float3 radiance = make_float3(gsRadDns.x, 
										  gsRadDns.y, 
										  gsRadDns.z);
			float alpha = gsRadDns.w;
			
			payload.accumColor += make_float3(1.0f - payload.accumAlpha) * radiance;
			payload.accumAlpha = clamp(payload.accumAlpha + alpha, 0.0f, 1.0f);
			payload.blockingRadiance = clamp(payload.blockingRadiance + alpha, 0.0f, 1.0f);
		}

		payload.accumColor += payload.directLight * (1.0f - payload.blockingRadiance);

		timeout += 1;
		if (timeout > TIMEOUT_ITERATIONS)
        	break;
	}

	float3 rgb = make_float3(payload.accumColor.x, payload.accumColor.y, payload.accumColor.z);
	writeOutputBuffer(rgb);
}

extern "C" __global__ void __miss__miss()
{
	if (getNextTraceState() == TraceMeshPass) {
		RayPayload* payload = getRayPayLoad();

		payload->currRayOrigin    = make_float3(0.0f);
		payload->currRayDirection = make_float3(0.0f);

		setNextTraceState(TraceLastGaussianPass);
	}
}

#define compareAndSwapHitPayloadValue(hit, i_id, i_distance)                      \
    {                                                                             \
        const float distance = __uint_as_float(optixGetPayload_##i_distance##()); \
        if (hit.distance < distance) {                                            \
            optixSetPayload_##i_distance##(__float_as_uint(hit.distance));        \
            const uint32_t id = optixGetPayload_##i_id##();                       \
            optixSetPayload_##i_id##(hit.particleId);                             \
            hit.distance   = distance;                                            \
            hit.particleId = id;                                                  \
        }                                                                         \
    }

extern "C" __global__ void __anyhit__anyhit()
{
	HitPayload hit = HitPayload{ optixGetInstanceId(), optixGetRayTmax() };
	if (hit.distance < __uint_as_float(optixGetPayload_13())) {
		compareAndSwapHitPayloadValue(hit, 0, 1);
		compareAndSwapHitPayloadValue(hit, 2, 3);
		compareAndSwapHitPayloadValue(hit, 4, 5);
		compareAndSwapHitPayloadValue(hit, 6, 7);
		compareAndSwapHitPayloadValue(hit, 8, 9);
		compareAndSwapHitPayloadValue(hit, 10, 11);
		compareAndSwapHitPayloadValue(hit, 12, 13);

		// ignore all inserted hits, expect if the last one
		if (__uint_as_float(optixGetPayload_13()) > optixGetRayTmax()) {
			optixIgnoreIntersection();
		}
	}
}

extern "C" __global__ void __closesthit__closesthit()
{
	RayPayload* payload = getRayPayLoad();
	unsigned int numBounces = payload->numBounces;
	unsigned int nextState = getNextTraceState();

	float  t_hit = optixGetRayTmax();
	float3 ray_o = optixGetWorldRayOrigin();
	float3 ray_d = optixGetWorldRayDirection();

	Mesh hitMesh  = params.d_meshes[optixGetInstanceId()];
	float3 normal = getBarycentricNormal(hitMesh);

	float3 newRayDirection = make_float3(0.0f);
	nextState = TraceGaussianPass;

	if (params.type == MIRROR)
		renderMirror(ray_d, normal, newRayDirection, numBounces);
	else if (params.type == NORMAL) {
		renderNormal(ray_o, ray_d, normal, t_hit, nextState, payload);
	}
	else if (params.type == GLASS) {
		renderGlass(ray_d, normal, newRayDirection, t_hit, numBounces);
	}

	payload->t_hit            = t_hit;
	payload->currRayOrigin    = ray_o + t_hit * ray_d;
	payload->currRayDirection = newRayDirection;
	payload->hitNormal        = normal;
	payload->numBounces       = numBounces;

	setNextTraceState(nextState);
}